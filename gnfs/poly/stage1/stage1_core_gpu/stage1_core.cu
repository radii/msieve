#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------
This source distribution is placed in the public domain by its author,
Jason Papadopoulos. You may use it for any purpose, free of charge,
without having to notify anyone. I disclaim any responsibility for any
errors.

Optionally, please be nice and tell me if you find this source to be
useful. Again optionally, if you add to the functionality present here
please consider making those additions public too, so that others may 
benefit from your work.	

$Id$
--------------------------------------------------------------------*/

#include "stage1_core.h"

#ifdef __cplusplus
extern "C" {
#endif

/*------------------------------------------------------------------------*/
__global__ void
sieve_kernel_trans_pp32_r32(uint32 *p_array, uint32 num_p, uint32 *start_roots,
			uint32 num_roots, uint32 *p_out, uint32 *roots_out,
			specialq_t *q_batch, uint32 num_specialq, 
			uint32 specialq_block, uint32 num_entries, 
			uint32 shift, uint32 num_aprog_vals)
{
	uint32 p, pp, pp_w, p_offset;
	uint32 specialq_start, specialq_end;
	uint32 q, qq_prod, qq_prod_offset, curr_offset, q_count;
	uint32 i, j, k, m, start_i, gcd, inv, curr_inv;
	uint32 qroot, newroot;
	specialq_t *curr_q;
	uint32 aprog_stride;

	p_offset = blockIdx.x * blockDim.x + threadIdx.x;
	if (p_offset >= num_p)
		return;

	p = p_array[p_offset];
	pp = p * p;
	pp_w = montmul32_w(pp);

	specialq_start = blockIdx.y * specialq_block;
	specialq_end = min(specialq_start + specialq_block, num_specialq);
	aprog_stride = num_entries * num_specialq;

	qq_prod_offset = specialq_start * num_entries + p_offset;
	curr_q = q_batch + specialq_start;

	q = j = 0;
	for (i = specialq_start; j == 0 && i < specialq_end; i++) {

		if (q != curr_q->p) {
			q = curr_q->p;
			gcd = gcd32(p, q);
			if (gcd == 1)
				j = qq_prod = curr_q->pp % pp;
		}

		roots_out[qq_prod_offset] = j;
		qq_prod_offset += num_entries;
		curr_q++;
	}
	if (j == 0)
		return;

	for (start_i = i - 1; i < specialq_end; i++) {

		if (q != curr_q->p) {
			q = curr_q->p;
			gcd = gcd32(p, q);

			if (gcd == 1)
				j = qq_prod = montmul32(qq_prod, 
						curr_q->pp % pp, 
						pp, pp_w);
			else
				j = 0;
		}

		roots_out[qq_prod_offset] = j;
		qq_prod_offset += num_entries;
		curr_q++;
	}

	inv = modinv32(qq_prod, pp);
	inv = montmul32(inv, montmul32_r(pp), pp, pp_w);
	qq_prod_offset -= num_entries;

	for (i--; i > start_i; i--) {

		uint32 curr_qq_prod = roots_out[qq_prod_offset];

		if (curr_qq_prod > 0)
			break;

		qq_prod_offset -= num_entries;
	}

	curr_offset = qq_prod_offset - num_entries;
	q = i;
	q_count = 1;

	for (i--; (int32)i >= (int32)start_i; 
			i--, curr_offset -= num_entries) {

		uint32 curr_qq_prod = roots_out[curr_offset];

		if (curr_qq_prod == 0) {
			continue;
		}
		else if (curr_qq_prod == qq_prod) {
			q_count++;
			continue;
		}

		curr_inv = montmul32(curr_qq_prod, inv, pp, pp_w);
		inv = montmul32(inv, q_batch[q].pp % pp, pp, pp_w);

		do {
			qroot = q_batch[q].root % pp;

			for (j = qq_prod_offset, k = p_offset, m = 0; 
						m < num_roots; 
						j += num_p, k += num_p, m++) {

				newroot = modsub32(start_roots[k], 
							qroot, pp);
				newroot = montmul32(newroot, curr_inv, 
							pp, pp_w);

				if (num_aprog_vals == 1) {
					if (newroot > pp / 2)
						newroot -= pp;

					p_out[j] = (q << shift) | p;
					roots_out[j] = newroot;
				}
				else {
					uint32 n;
					uint32 r = j;

					newroot -= pp * (num_aprog_vals / 2);

					for (n = 0; n < num_aprog_vals; n++) {
						p_out[r] = (q << shift) | p;
						roots_out[r] = newroot;
						r += aprog_stride;
						newroot += pp;
					}
				}
			}

			q--;
			qq_prod_offset -= num_entries;
		} while (--q_count);

		q = i;
		q_count = 1;
		qq_prod = curr_qq_prod;
		qq_prod_offset = curr_offset;
	}

	curr_inv = inv;
	while ((int32)q >= (int32)start_i) {

		qroot = q_batch[q].root % pp;

		for (j = qq_prod_offset, k = p_offset, m = 0; 
					m < num_roots; 
					j += num_p, k += num_p, m++) {

			newroot = modsub32(start_roots[k], 
						qroot, pp);
			newroot = montmul32(newroot, curr_inv, 
						pp, pp_w);

			if (num_aprog_vals == 1) {
				if (newroot > pp / 2)
					newroot -= pp;

				p_out[j] = (q << shift) | p;
				roots_out[j] = newroot;
			}
			else {
				uint32 n;
				uint32 r = j;

				newroot -= pp * (num_aprog_vals / 2);

				for (n = 0; n < num_aprog_vals; n++) {
					p_out[r] = (q << shift) | p;
					roots_out[r] = newroot;
					r += aprog_stride;
					newroot += pp;
				}
			}
		}

		q--;
		qq_prod_offset -= num_entries;
	}
}

/*------------------------------------------------------------------------*/
__global__ void
sieve_kernel_trans_pp32_r64(uint32 *p_array, uint32 num_p, uint32 *start_roots,
			uint32 num_roots, uint32 *p_out, uint64 *roots_out,
			specialq_t *q_batch, uint32 num_specialq, 
			uint32 specialq_block, uint32 num_entries, 
			uint32 shift, uint32 num_aprog_vals)
{
	uint32 p, pp, pp_w, p_offset;
	uint32 specialq_start, specialq_end;
	uint32 q, qq_prod, qq_prod_offset, curr_offset, q_count;
	uint32 i, j, k, m, start_i, gcd, inv, curr_inv;
	uint32 qroot, newroot;
	specialq_t *curr_q;
	uint32 aprog_stride;

	p_offset = blockIdx.x * blockDim.x + threadIdx.x;
	if (p_offset >= num_p)
		return;

	p = p_array[p_offset];
	pp = p * p;
	pp_w = montmul32_w(pp);

	specialq_start = blockIdx.y * specialq_block;
	specialq_end = min(specialq_start + specialq_block, num_specialq);
	aprog_stride = num_entries * num_specialq;

	qq_prod_offset = specialq_start * num_entries + p_offset;
	curr_q = q_batch + specialq_start;

	q = j = 0;
	for (i = specialq_start; j == 0 && i < specialq_end; i++) {

		if (q != curr_q->p) {
			q = curr_q->p;
			gcd = gcd32(p, q);
			if (gcd == 1)
				j = qq_prod = curr_q->pp % pp;
		}

		p_out[qq_prod_offset] = j;
		qq_prod_offset += num_entries;
		curr_q++;
	}
	if (j == 0)
		return;

	for (start_i = i - 1; i < specialq_end; i++) {

		if (q != curr_q->p) {
			q = curr_q->p;
			gcd = gcd32(p, q);

			if (gcd == 1)
				j = qq_prod = montmul32(qq_prod, 
						curr_q->pp % pp, 
						pp, pp_w);
			else
				j = 0;
		}

		p_out[qq_prod_offset] = j;
		qq_prod_offset += num_entries;
		curr_q++;
	}

	inv = modinv32(qq_prod, pp);
	inv = montmul32(inv, montmul32_r(pp), pp, pp_w);
	qq_prod_offset -= num_entries;

	for (i--; i > start_i; i--) {

		uint32 curr_qq_prod = p_out[qq_prod_offset];

		if (curr_qq_prod > 0)
			break;

		qq_prod_offset -= num_entries;
	}

	curr_offset = qq_prod_offset - num_entries;
	q = i;
	q_count = 1;

	for (i--; (int32)i >= (int32)start_i; 
			i--, curr_offset -= num_entries) {

		uint32 curr_qq_prod = p_out[curr_offset];

		if (curr_qq_prod == 0) {
			continue;
		}
		else if (curr_qq_prod == qq_prod) {
			q_count++;
			continue;
		}

		curr_inv = montmul32(curr_qq_prod, inv, pp, pp_w);
		inv = montmul32(inv, q_batch[q].pp % pp, pp, pp_w);

		do {
			qroot = q_batch[q].root % pp;

			for (j = qq_prod_offset, k = p_offset, m = 0; 
						m < num_roots; 
						j += num_p, k += num_p, m++) {

				newroot = modsub32(start_roots[k], 
							qroot, pp);
				newroot = montmul32(newroot, curr_inv, 
							pp, pp_w);

				if (num_aprog_vals == 1) {
					if (newroot > pp / 2)
						newroot -= pp;

					p_out[j] = (q << shift) | p;
					roots_out[j] = newroot;
				}
				else {
					uint64 newroot64 = newroot;
					uint32 n;
					uint32 r = j;

					newroot64 -= (uint64)pp * 
							(num_aprog_vals / 2);

					for (n = 0; n < num_aprog_vals; n++) {
						p_out[r] = (q << shift) | p;
						roots_out[r] = newroot64;
						r += aprog_stride;
						newroot64 += pp;
					}
				}
			}

			q--;
			qq_prod_offset -= num_entries;
		} while (--q_count);

		q = i;
		q_count = 1;
		qq_prod = curr_qq_prod;
		qq_prod_offset = curr_offset;
	}

	curr_inv = inv;
	while ((int32)q >= (int32)start_i) {

		qroot = q_batch[q].root % pp;

		for (j = qq_prod_offset, k = p_offset, m = 0; 
					m < num_roots; 
					j += num_p, k += num_p, m++) {

			newroot = modsub32(start_roots[k], 
						qroot, pp);
			newroot = montmul32(newroot, curr_inv, 
						pp, pp_w);

			if (num_aprog_vals == 1) {
				if (newroot > pp / 2)
					newroot -= pp;

				p_out[j] = (q << shift) | p;
				roots_out[j] = newroot;
			}
			else {
				uint64 newroot64 = newroot;
				uint32 n;
				uint32 r = j;

				newroot64 -= (uint64)pp * 
						(num_aprog_vals / 2);

				for (n = 0; n < num_aprog_vals; n++) {
					p_out[r] = (q << shift) | p;
					roots_out[r] = newroot64;
					r += aprog_stride;
					newroot64 += pp;
				}
			}
		}

		q--;
		qq_prod_offset -= num_entries;
	}
}

/*------------------------------------------------------------------------*/
__global__ void
sieve_kernel_trans_pp64_r64(uint32 *p_array, uint32 num_p, uint64 *start_roots,
			uint32 num_roots, uint32 *p_out, int64 *roots_out,
			specialq_t *q_batch, uint32 num_specialq, 
			uint32 specialq_block, uint32 num_entries, 
			uint32 shift, uint32 num_aprog_vals)
{
	uint32 p, pp_w, p_offset;
	uint64 pp, qq_prod, qroot, newroot, write_val;
	uint32 specialq_start, specialq_end;
	uint32 q, qq_prod_offset, curr_offset, q_count;
	uint32 i, j, k, m, start_i, gcd; 
	uint64 inv, curr_inv;
	specialq_t *curr_q;
	uint32 aprog_stride;

	p_offset = blockIdx.x * blockDim.x + threadIdx.x;
	if (p_offset >= num_p)
		return;

	p = p_array[p_offset];
	pp = (uint64)p * p;
	pp_w = montmul32_w(pp);

	specialq_start = blockIdx.y * specialq_block;
	specialq_end = min(specialq_start + specialq_block, num_specialq);
	aprog_stride = num_entries * num_specialq;

	qq_prod_offset = specialq_start * num_entries + p_offset;
	curr_q = q_batch + specialq_start;

	q = 0;
	write_val = 0;
	for (i = specialq_start; write_val == 0 && i < specialq_end; i++) {

		if (q != curr_q->p) {
			q = curr_q->p;
			gcd = gcd32(p, q);
			if (gcd == 1)
				write_val = qq_prod = curr_q->pp % pp;
		}

		roots_out[qq_prod_offset] = write_val;
		qq_prod_offset += num_entries;
		curr_q++;
	}
	if (write_val == 0)
		return;

	for (start_i = i - 1; i < specialq_end; i++) {

		if (q != curr_q->p) {
			q = curr_q->p;
			gcd = gcd32(p, q);

			if (gcd == 1)
				write_val = qq_prod = montmul64(qq_prod, 
						curr_q->pp % pp, 
						pp, pp_w);
			else
				write_val = 0;
		}

		roots_out[qq_prod_offset] = write_val;
		qq_prod_offset += num_entries;
		curr_q++;
	}

	inv = modinv64(qq_prod, pp);
	inv = montmul64(inv, montmul64_r(pp, pp_w), pp, pp_w);
	qq_prod_offset -= num_entries;

	for (i--; i > start_i; i--) {

		uint32 curr_qq_prod = roots_out[qq_prod_offset];

		if (curr_qq_prod > 0)
			break;

		qq_prod_offset -= num_entries;
	}

	curr_offset = qq_prod_offset - num_entries;
	q = i;
	q_count = 1;

	for (i--; (int32)i >= (int32)start_i; 
			i--, curr_offset -= num_entries) {

		uint64 curr_qq_prod = roots_out[curr_offset];

		if (curr_qq_prod == 0) {
			continue;
		}
		else if (curr_qq_prod == qq_prod) {
			q_count++;
			continue;
		}

		curr_inv = montmul64(curr_qq_prod, inv, pp, pp_w);
		inv = montmul64(inv, q_batch[q].pp % pp, pp, pp_w);

		do {
			qroot = q_batch[q].root % pp;

			for (j = qq_prod_offset, k = p_offset, m = 0; 
						m < num_roots; 
						j += num_p, k += num_p, m++) {

				newroot = modsub64(start_roots[k], 
							qroot, pp);
				newroot = montmul64(newroot, curr_inv, 
							pp, pp_w);

				if (num_aprog_vals == 1) {
					if (newroot > pp / 2)
						newroot -= pp;

					p_out[j] = (q << shift) | p;
					roots_out[j] = newroot;
				}
				else {
					uint32 n;
					uint32 r = j;

					newroot -= pp * (num_aprog_vals / 2);

					for (n = 0; n < num_aprog_vals; n++) {
						p_out[r] = (q << shift) | p;
						roots_out[r] = newroot;
						r += aprog_stride;
						newroot += pp;
					}
				}
			}

			q--;
			qq_prod_offset -= num_entries;
		} while (--q_count);

		q = i;
		q_count = 1;
		qq_prod = curr_qq_prod;
		qq_prod_offset = curr_offset;
	}

	curr_inv = inv;
	while ((int32)q >= (int32)start_i) {

		qroot = q_batch[q].root % pp;

		for (j = qq_prod_offset, k = p_offset, m = 0; 
					m < num_roots; 
					j += num_p, k += num_p, m++) {

			newroot = modsub64(start_roots[k], 
						qroot, pp);
			newroot = montmul64(newroot, curr_inv, 
						pp, pp_w);

			if (num_aprog_vals == 1) {
				if (newroot > pp / 2)
					newroot -= pp;

				p_out[j] = (q << shift) | p;
				roots_out[j] = newroot;
			}
			else {
				uint32 n;
				uint32 r = j;

				newroot -= pp * (num_aprog_vals / 2);

				for (n = 0; n < num_aprog_vals; n++) {
					p_out[r] = (q << shift) | p;
					roots_out[r] = newroot;
					r += aprog_stride;
					newroot += pp;
				}
			}
		}

		q--;
		qq_prod_offset -= num_entries;
	}
}

/*------------------------------------------------------------------------*/
__device__ void
store_hit(found_t *found_array, uint32 found_array_size,
		uint32 p1, uint32 p2,
		int64 root, specialq_t *q)
{
	/* don't use atomicInc because we don't want
	   wraparound to occur */

	uint32 index = atomicAdd(&found_array[0].p1, 1);

	if (index < found_array_size - 1) {

		found_t *f = found_array + index + 1;

		f->p1 = p1;
		f->p2 = p2;
		f->q = q->p;
		f->qroot = q->root;
		f->offset = root;
	}
}

/*------------------------------------------------------------------------*/
__global__ void
sieve_kernel_final_32(uint32 *p_array, int32 *roots, uint32 p_array_size,
			specialq_t * q_batch, found_t *found_array, 
			uint32 shift)
{
	uint32 i, j;
	uint32 num_threads = gridDim.x * blockDim.x;
	uint32 my_threadid = blockIdx.x * blockDim.x + threadIdx.x;
	uint32 mask = (1 << shift) - 1;

	for (i = my_threadid; i < p_array_size - 1; i += num_threads) {

		int32 root1 = roots[i];
		uint32 p1 = p_array[i];

		if (root1 == 0)
			continue;

		for (j = i + 1; j < p_array_size; j++) {
			int32 root2 = roots[j];
			uint32 p2 = p_array[j];

			if (root1 != root2)
				break;

			if ((p1 >> shift) == (p2 >> shift) &&
			    gcd32( (p1 & mask), (p2 & mask) ) == 1) {

				store_hit(found_array, FOUND_ARRAY_SIZE,
						p1 & mask, p2 & mask, 
						(int64)root1,
						q_batch + (p1 >> shift));
			}
		}
	}
}

/*------------------------------------------------------------------------*/
__global__ void
sieve_kernel_final_64(uint32 *p_array, int64 *roots, uint32 p_array_size,
			specialq_t * q_batch, found_t *found_array, 
			uint32 shift)
{
	uint32 i, j;
	uint32 num_threads = gridDim.x * blockDim.x;
	uint32 my_threadid = blockIdx.x * blockDim.x + threadIdx.x;
	uint32 mask = (1 << shift) - 1;

	for (i = my_threadid; i < p_array_size - 1; i += num_threads) {

		int64 root1 = roots[i];
		uint32 p1 = p_array[i];

		if (root1 == 0)
			continue;

		for (j = i + 1; j < p_array_size; j++) {
			int64 root2 = roots[j];
			uint32 p2 = p_array[j];

			if (root1 != root2)
				break;

			if ((p1 >> shift) == (p2 >> shift) &&
			    gcd32( (p1 & mask), (p2 & mask) ) == 1) {

				store_hit(found_array, FOUND_ARRAY_SIZE,
						p1 & mask, p2 & mask, root1,
						q_batch + (p1 >> shift));
			}
		}
	}
}

#ifdef __cplusplus
}
#endif
